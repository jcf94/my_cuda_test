/* ***********************************************
MYID	: Chen Fan
LANG	: NVCC
PROG	: GMATRIX
************************************************ */

#include "gmatrix.h"

#include <stdio.h>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
using namespace std;


#define check_cuda_rt(x) do { hipError_t error__ = x; if (error__ != hipSuccess) { printf("Code: %s, Description: %s", hipGetErrorName(error__), hipGetErrorString(error__)); exit(EXIT_FAILURE); } } while(false)

#define ADDR(x) ((void**)&(x))

#include "gemm.h"

template<typename T>
gmatrix<T>::gmatrix(int x, int y)
    : matrix<T>(x, y)
{
    printf("gmatrix Normal Construct\n");

    int size = sizeof(T)*x*y;
    printf("Size: %d\n", size);

    check_cuda_rt(hipMalloc(ADDR(_gpu_data), size));
    printf("gpu_data: %p\n", _gpu_data);
    printf("cpu_data: %p\n", matrix<T>::_data);
}

template<typename T>
gmatrix<T>::gmatrix(const gmatrix& b) // copy construct
{
    printf("Copy Construct\n");

    matrix<T>::_x = b._x;
    matrix<T>::_y = b._y;
    matrix<T>::_data = new T[b._x*b._y];
    int data_size = sizeof(T)*x()*y();
    memcpy(data(), b._data, data_size);

    check_cuda_rt(hipMalloc(ADDR(_gpu_data), data_size));
    check_cuda_rt(hipMemcpy(_gpu_data, b._gpu_data, data_size, hipMemcpyDeviceToDevice));
}

template<typename T>
gmatrix<T>& gmatrix<T>::operator=(const gmatrix& b) // copy assign
{
    printf("Copy Assign\n");

    if (this != &b)
    {
        int data_size = sizeof(T)*b._x*b._y;
        if (matrix<T>::_x!=b._x || matrix<T>::_y!=b._y)
        {
            matrix<T>::_x = b._x;
            matrix<T>::_y = b._y;
            delete[] matrix<T>::_data;
            matrix<T>::_data = new T[matrix<T>::_x*matrix<T>::_y];
            check_cuda_rt(hipFree(_gpu_data));
            check_cuda_rt(hipMalloc(ADDR(_gpu_data), data_size));
        }
        memcpy(matrix<T>::_data, b._data, data_size);
        check_cuda_rt(hipMemcpy(_gpu_data, b._gpu_data, data_size, hipMemcpyDeviceToDevice));
    }
    return *this;
}

template<typename T>
gmatrix<T>::gmatrix(gmatrix&& b) // move construct
{
    matrix<T>::_x = b._x;
    matrix<T>::_y = b._y;
    matrix<T>::_data = b._data;
    _gpu_data = b._gpu_data;

    b._data = NULL;
    b._gpu_data = NULL;
    b._x = 0;
    b._y = 0;
}

template<typename T>
gmatrix<T>& gmatrix<T>::operator=(gmatrix&& b) // move assign
{
    if (this != &b)
    {
        delete[] matrix<T>::_data;
        check_cuda_rt(hipFree(_gpu_data));

        matrix<T>::_x = b._x;
        matrix<T>::_y = b._y;
        matrix<T>::_data = b._data;
        _gpu_data = b._gpu_data;

        b._x = 0;
        b._y = 0;
        b._data = NULL;
        b._gpu_data = NULL;
    }
    return *this;
}

template<typename T>
gmatrix<T>::~gmatrix()
{
    printf("Gmatrix Delete\n");
    printf("gpu_data: %p\n", _gpu_data);
    cout << hipFree(_gpu_data) << endl;
    printf("Gmatrix Delete Complete\n");
}

template<typename T>
gmatrix<T> gmatrix<T>::operator+(const gmatrix<T>& b)
{
    if (matrix<T>::_x != b._x || matrix<T>::_y != b._y)
    {
        printf("Shape Error!");
        return NULL;
    }

    gmatrix<T> c(matrix<T>::_x, matrix<T>::_y);

    int block_dim = 512;
    int grid_dim = (matrix<T>::_x*matrix<T>::_y + block_dim - 1) / block_dim;
    add_kernel<T><<<grid_dim, block_dim>>>(c._gpu_data, _gpu_data, b._gpu_data,
        matrix<T>::_x, matrix<T>::_y);
    hipDeviceSynchronize();

    return c;
}

template<typename T>
gmatrix<T> gmatrix<T>::operator*(const gmatrix<T>& b)
{
    if (matrix<T>::_y != b._x)
    {
        printf("Shape Error!");
        return NULL;
    }

    gmatrix<T> c(matrix<T>::_x, b._y);

    dim3 block_dim(TILE_WIDTH, TILE_WIDTH);
    dim3 grid_dim((matrix<T>::_x + TILE_WIDTH - 1) / TILE_WIDTH, (b._y + TILE_WIDTH - 1) / TILE_WIDTH);
    mul_kernel<T><<<grid_dim, block_dim>>>(c._gpu_data, _gpu_data, b._gpu_data,
        matrix<T>::_x, matrix<T>::_y, b._y);
    hipDeviceSynchronize();

    return c;
}

#include <omp.h>

template<typename T>
gmatrix<T> gmatrix<T>::operator*(matrix<T> b)
{
    if (y() != b.x())
    {
        printf("Shape Error!");
        return NULL;
    }

    gmatrix<T> c(x(), b.y());

    for (int i=0;i<x();i++)
    #pragma omp parallel for
    for (int j=0;j<b.y();j++)
    {
        T temp = 0;
        for (int k=0;k<y();k++)
        temp += data()[index(i, k)]*b[k][j];
        c[i][j] = temp;
    }

    return c;
}

template<typename T>
void gmatrix<T>::hTod()
{
    printf("cudamemcpy\n");
    int size = sizeof(T)*x()*y();
    printf("Size: %d\n", size);
    printf("gpu_data: %p\n", _gpu_data);
    printf("cpu_data: %p\n", matrix<T>::_data);
    check_cuda_rt(hipMemcpy(_gpu_data, matrix<T>::_data,
        size, hipMemcpyHostToDevice));
    printf("cudamemcpy_end\n");
}

template<typename T>
void gmatrix<T>::dToh()
{
    check_cuda_rt(hipMemcpy(matrix<T>::_data, _gpu_data,
        sizeof(T)*matrix<T>::_x*matrix<T>::_y,
        hipMemcpyDeviceToHost));
}

template<typename T>
bool gmatrix<T>::equal(gmatrix<T> b)
{
    if (x()!=b.x() || y()!=b.y()) return false;
    for (int i=0;i<x();i++)
    for (int j=0;j<y();j++)
    if (data()[index(i, j)] != b[i][j]) return false;
    return true;
}

template class gmatrix<int>;
template class gmatrix<float>;
template class gmatrix<double>;