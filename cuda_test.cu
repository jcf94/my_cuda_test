/* ***********************************************
MYID	: Chen Fan
LANG	: G++
PROG	: CUDA_TEST
************************************************ */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_test.h"

void check_device()
{
    printf("Starting...\n\n");
    printf(" CUDA Device Query (Runtime API) version (CUDART static linking)\n\n");

    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
    {
        printf("There are no available device(s) that support CUDA\n");
    }
    else
    {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }
}